#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>

#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <hip/hip_runtime_api.h>


#include "writeToCSVFileHeader.h"
#include "userInputHeader.h"

void printArray(int *elements);

int stepOneThreadCount;
int blockCount;
int allOtherStepsThreadCount;
int elementCount;

int executionCount;


bool isSorted(int *elements){

	bool sorted = true;
	for (int i = 0; i < (elementCount - 1); ++i){
		if (elements[i] > elements[i + 1]){
			sorted = false;
		}
	}
	return sorted;
}

double getElapsedTime(clock_t start, clock_t stop)
{
	double elapsed = ((double)(stop - start)) / CLOCKS_PER_SEC;
	printf("Elapsed time: %.3fs\n", elapsed);

	return elapsed;
}


const int randMax = 10000;

void createUnsortedArray(int* elements){

	for (int i = 0; i < elementCount; ++i){
		elements[i] = rand() % randMax - rand() % 5;
	}

}

struct arrays
{
	int* evenArray;
	int* oddArray;
};

arrays splitArray(int* origionalElements, arrays evenOddArrays){

	int *evenCountPtr = evenOddArrays.evenArray;
	int *oddCountPtr = evenOddArrays.oddArray;
	int count = 0;
	for (int i = 0; i < elementCount; i++){
	
		if (i % 2 == 0){

			memcpy(evenCountPtr, origionalElements, sizeof(int));
		
			evenCountPtr++;
		}
		else{
			memcpy(oddCountPtr, origionalElements, sizeof(int));

			oddCountPtr++;
		}
		origionalElements++;
	}

	return evenOddArrays;
}

//Merger even and odd arrays into origional arrays
void mergeArrays(arrays evenOddArrays,  int* origionalElemens){

	int *evenCountPtr = evenOddArrays.evenArray;
	int *oddCountPtr = evenOddArrays.oddArray;
	int count = 0;

	for (int i = 0; i < elementCount; i++){
		
		if (i % 2 == 0){

			//Destination, Source, size
			memcpy(origionalElemens, evenCountPtr, sizeof(int));
			evenCountPtr++;
		}
		else{
			memcpy(origionalElemens, oddCountPtr, sizeof(int));
			oddCountPtr++;
		}


		origionalElemens++;
	}

}

bool checkEvenOddArrays(arrays evenOddArrays,  int* origionaArray){

	int evenCount = 0;
	int oddCount = 0;

	bool elementSplitCorrectly = true;

	for (int i = 0; i < elementCount; i++){
		
		if (i % 2 == 0){

			if (origionaArray[i] != evenOddArrays.evenArray[evenCount]){

				elementSplitCorrectly = false;
			}
			evenCount++;
		}

		else{

			if (origionaArray[i] != evenOddArrays.oddArray[oddCount]){

				elementSplitCorrectly = false;
			}
			oddCount++;
		}
	}

	return elementSplitCorrectly;
}

__global__ void bitonicSortAllOtherStepsSingleThreaded(int *deviceElements, int step, int phase, int compExchSize, int fullArraySize)
{
	unsigned int  halfstep, halfphase, secondIndex;

	int firstIndex = threadIdx.x + blockDim.x * blockIdx.x;

	for (int count = 0; count < fullArraySize / 2; count++){
		halfstep = step / 2;
		halfphase = phase / 2;
		secondIndex = firstIndex^halfstep;

		if ((secondIndex) > firstIndex) {
			if ((firstIndex&halfphase) == 0) {

				if (deviceElements[firstIndex] > deviceElements[secondIndex]) {
					int temp = deviceElements[firstIndex];
					deviceElements[firstIndex] = deviceElements[secondIndex];
					deviceElements[secondIndex] = temp;
				}
			}
			if ((firstIndex&halfphase) != 0) {

				if (deviceElements[firstIndex] < deviceElements[secondIndex]) {
					int temp = deviceElements[firstIndex];
					deviceElements[firstIndex] = deviceElements[secondIndex];
					deviceElements[secondIndex] = temp;
				}
			}
		}
		firstIndex++;
	}
}

/*	As we're complying with the origional model but with arrays reduced in size by two 
	here we'll just divide firstIndex and secondIndex by two to get our required values.
	We're also going to go though all the compaire/exchange operations that would normally be carried out in this step.
*/
__global__ void bitonicSortAllOtherSteps(int *deviceElements, int step, int phase, int compExchSize, int fullArraySize)
{
	unsigned int  halfstep, halfphase, secondIndex;

	int firstIndex = threadIdx.x + blockDim.x * blockIdx.x;

	halfstep = step/2;
	halfphase = phase/2;
	secondIndex = firstIndex^halfstep;

	if ((secondIndex) > firstIndex) {
		if ((firstIndex&halfphase) == 0) {
			if (deviceElements[firstIndex] > deviceElements[secondIndex]) {
				int temp = deviceElements[firstIndex];
				deviceElements[firstIndex] = deviceElements[secondIndex];
				deviceElements[secondIndex] = temp;
				}
		}
		if ((firstIndex&halfphase) != 0) {

			if (deviceElements[firstIndex] < deviceElements[secondIndex]) {
				int temp = deviceElements[firstIndex];
				deviceElements[firstIndex] = deviceElements[secondIndex];
				deviceElements[secondIndex] = temp;
			}
		}

	}
}

__global__ void bitonicSortFirstStep(int *deviceElements, int step, int phase)
{
	unsigned int firstIndex, secondIndex; 
	firstIndex = threadIdx.x + blockDim.x * blockIdx.x;

	secondIndex = firstIndex^step;

	if ((secondIndex)>firstIndex) {
		if ((firstIndex& phase) == 0) {

			if (deviceElements[firstIndex]>deviceElements[secondIndex]) {
				int temp = deviceElements[firstIndex];
				deviceElements[firstIndex] = deviceElements[secondIndex];
				deviceElements[secondIndex] = temp;
			}
		}
		if ((firstIndex&phase) != 0) {

			if (deviceElements[firstIndex]<deviceElements[secondIndex]) {

				int temp = deviceElements[firstIndex];
				deviceElements[firstIndex] = deviceElements[secondIndex];
				deviceElements[secondIndex] = temp;
			}
		}
	}
}


//Launcher function for our instep inplmentation of bitonic sort
void bitonic_sort(int *values)
{
	int *deviceElements;
	int *deviceEvenArray;
	int *deviceOddArray;
	size_t size = elementCount * sizeof(int);

	size_t evenOddSize = elementCount/2 * sizeof(int);

	arrays evenOddArrays;


	//Allocate half of element size to each of the odd and even arrays
	evenOddArrays.evenArray = (int*)malloc((elementCount)* sizeof(int));
	evenOddArrays.oddArray = (int*)malloc((elementCount)* sizeof(int));

	hipMalloc((void**)&deviceElements, size);
	hipMalloc((void**)&deviceEvenArray, evenOddSize);
	hipMalloc((void**)&deviceOddArray, evenOddSize);

	dim3 blocks(blockCount, 1);    
	dim3 stepOneThreads(stepOneThreadCount, 1);

	dim3 allOtherStepThreads(allOtherStepsThreadCount, 1);

	int compExchCount = (elementCount / 4);

	int step, phase;

	for (phase = 2; phase <= elementCount; phase <<= 1) {
		evenOddArrays = splitArray(values, evenOddArrays);

		hipMemcpy(deviceEvenArray, evenOddArrays.evenArray, evenOddSize, hipMemcpyHostToDevice);
		hipMemcpy(deviceOddArray, evenOddArrays.oddArray, evenOddSize, hipMemcpyHostToDevice);

		for (step = phase >> 1; step > 0 ; step = step >> 1) {

			if (step != 1){
			
				//Even and odd arrays to kernals 			
				bitonicSortAllOtherSteps << <blocks, allOtherStepThreads >> >(deviceEvenArray, step, phase, compExchCount, elementCount);
				bitonicSortAllOtherSteps << <blocks, allOtherStepThreads >> >(deviceOddArray, step, phase, compExchCount, elementCount);
				
			}

			//The last step, so copy back the sorted even odd arrays, merge them into the origional element array copy that to memory then sort it 
			else{
			
				hipMemcpy(evenOddArrays.evenArray, deviceEvenArray, evenOddSize, hipMemcpyDeviceToHost);
				hipMemcpy(evenOddArrays.oddArray, deviceOddArray, evenOddSize, hipMemcpyDeviceToHost);

				mergeArrays(evenOddArrays, values);
	
				hipMemcpy(deviceElements, values, size, hipMemcpyHostToDevice);
				bitonicSortFirstStep << <blocks, stepOneThreads >> >(deviceElements, step, phase);
	
				hipMemcpy(values, deviceElements, size, hipMemcpyDeviceToHost);

			}
		}
	}

	hipFree(deviceElements);
	hipFree(deviceEvenArray);
	hipFree(deviceOddArray);

}

void preExecution(){

	int values[7];
	values[0] = 10;
	values[1] = 13;
	values[2] = 9;
	values[3] = 18;
	values[4] = 26;
	values[4] = 100;
	values[6] = 3;

	bitonic_sort(values);
}

int main(void)
{

	executionCount = getMaxProcessCount();
	int fixedExecutionCount = executionCount;

	preExecution();

	bool runSort = true;

	//Pointers to store our results that we're writing to CSV files, allocate space entered buy the user
	int* threadCounts = (int*)malloc(executionCount*sizeof(int));
	int* allBlocks = (int*)malloc(executionCount*sizeof(int));;
	double* timeResults = (double*)malloc(executionCount*sizeof(double));;
	char* arrayStates = (char*)malloc(executionCount*sizeof(char));

	double time;
	clock_t start, stop;
	//Counter so we can assine values to the array in the execution loop

	while (runSort && executionCount != 0){

		runSort = runSortAgain();

		//Get thread, blocks and  element count

		//Get total elements and suggested block thread configurations
		blockAndThreadCounts inputCountandSuggestedThreadBlockCount;
		inputCountandSuggestedThreadBlockCount = getElementCounts();
		elementCount = inputCountandSuggestedThreadBlockCount.elementCount;

		//wirte possible thread and block configurations to text file
		printf("Writing suggested block thread configuration...");
		writeSuggestedBlockThreadConfigToCsv(inputCountandSuggestedThreadBlockCount.threadCounts,
			inputCountandSuggestedThreadBlockCount.blockCounts,
			inputCountandSuggestedThreadBlockCount.combinationsCount
			);
		printf("Done \n");

		//Get block count and thread count and thena assign half that thread count for all other steps
	    blockCount = getBlockCount();
		stepOneThreadCount = getThreadCount();
		allOtherStepsThreadCount = stepOneThreadCount / 2;


		//Malloc array, add values to it and write unsorted array to csv file
		int* values = (int*)malloc(elementCount*sizeof(int));
		createUnsortedArray(values);
		writeBlockElementCsvFile(values, "preSorted", stepOneThreadCount, blockCount);

		//Do Sort and time it
		start = clock();
		bitonic_sort(values);
		stop = clock();

		time = getElapsedTime(start, stop);

		char* arrayState;
		char arrayStateChar;

		if (isSorted(values)){

			printf("Is Sorted \n");
			arrayState = "sorted";
			arrayStateChar = 's';
		}
		else{

			printf("Not Sorted \n");
			arrayState = "unsorted";
			arrayStateChar = 'u';
		}

		writeBlockElementCsvFile(values, arrayState, stepOneThreadCount, blockCount);

		//Allocate results values to pointers 
		*threadCounts = stepOneThreadCount;
		*allBlocks = blockCount;
		*timeResults = time;
		*arrayStates = arrayStateChar;

		//Increment Result pointers
		threadCounts++;
		allBlocks++;
		timeResults++;
		arrayStates++;

		free(values);

		//Check again for user input

		executionCount--;
	}

	printf("Execution ended. Writing results to C:\BitonicSortArrayCSVFiles /n");

	writeSortResultsToCsv(timeResults, "PartitionedArray2DBitonicSort", arrayStates, threadCounts, allBlocks, fixedExecutionCount);

	getchar();

}